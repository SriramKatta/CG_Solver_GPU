#include "hip/hip_runtime.h"
// V1 added prefetching to all kernls to page fault data transfer between cpu and gpu
#include "cg-util.h"

#include "cuda-util.h"

#include <nvtx3/nvtx3.hpp>

template <typename tpe>
__global__ void cgAp(const tpe *const __restrict__ p, tpe *__restrict__ ap,
                     const size_t nx, const size_t ny) {
  size_t gridStartX = blockIdx.x * blockDim.x + threadIdx.x + 1;
  size_t gridStrideX = gridDim.x * blockDim.x;
  size_t gridStartY = blockIdx.y * blockDim.y + threadIdx.y + 1;
  size_t gridStrideY = gridDim.y * blockDim.y;

  for (size_t j = gridStartY; j < ny - 1; j += gridStrideY)
    for (size_t i = gridStartX; i < nx - 1; i += gridStrideX) {
      ap[j * nx + i] =
        4 * p[j * nx + i] - (p[j * nx + i - 1] + p[j * nx + i + 1] +
                             p[(j - 1) * nx + i] + p[(j + 1) * nx + i]);
    }
}

template <typename tpe>
__global__ void cgUpdateSol(const tpe *const __restrict__ p,
                            tpe *__restrict__ u, const tpe alpha,
                            const size_t nx, const size_t ny) {
  size_t gridStartX = blockIdx.x * blockDim.x + threadIdx.x + 1;
  size_t gridStrideX = gridDim.x * blockDim.x;
  size_t gridStartY = blockIdx.y * blockDim.y + threadIdx.y + 1;
  size_t gridStrideY = gridDim.y * blockDim.y;

  for (size_t j = gridStartY; j < ny - 1; j += gridStrideY)
    for (size_t i = gridStartX; i < nx - 1; i += gridStrideX) {
      u[j * nx + i] += alpha * p[j * nx + i];
    }
}

template <typename tpe>
__global__ void cgUpdateRes(const tpe *const __restrict__ ap,
                            tpe *__restrict__ res, const tpe alpha,
                            const size_t nx, const size_t ny) {
  size_t gridStartX = blockIdx.x * blockDim.x + threadIdx.x + 1;
  size_t gridStrideX = gridDim.x * blockDim.x;
  size_t gridStartY = blockIdx.y * blockDim.y + threadIdx.y + 1;
  size_t gridStrideY = gridDim.y * blockDim.y;

  for (size_t j = gridStartY; j < ny - 1; j += gridStrideY)
    for (size_t i = gridStartX; i < nx - 1; i += gridStrideX) {
      res[j * nx + i] = res[j * nx + i] - alpha * ap[j * nx + i];
    }
}

template <typename tpe>
__global__ void cgUpdateP(tpe beta, const tpe *const __restrict__ res,
                          tpe *__restrict__ p, size_t nx, size_t ny) {
  size_t gridStartX = blockIdx.x * blockDim.x + threadIdx.x + 1;
  size_t gridStrideX = gridDim.x * blockDim.x;
  size_t gridStartY = blockIdx.y * blockDim.y + threadIdx.y + 1;
  size_t gridStrideY = gridDim.y * blockDim.y;

  for (size_t j = gridStartY; j < ny - 1; j += gridStrideY)
    for (size_t i = gridStartX; i < nx - 1; i += gridStrideX) {
      p[j * nx + i] = res[j * nx + i] + beta * p[j * nx + i];
    }
}

template <typename tpe>
__global__ void residual_initp(tpe *__restrict__ res, tpe *__restrict__ p,
                               const tpe *const __restrict__ rhs,
                               const tpe *const __restrict__ u, size_t nx,
                               size_t ny) {

  size_t gridStartX = blockIdx.x * blockDim.x + threadIdx.x + 1;
  size_t gridStrideX = gridDim.x * blockDim.x;
  size_t gridStartY = blockIdx.y * blockDim.y + threadIdx.y + 1;
  size_t gridStrideY = gridDim.y * blockDim.y;

  for (size_t j = gridStartY; j < ny - 1; j += gridStrideY)
    for (size_t i = gridStartX; i < nx - 1; i += gridStrideX) {
      auto temp =
        rhs[j * nx + i] -
        (4 * u[j * nx + i] - (u[j * nx + i - 1] + u[j * nx + i + 1] +
                              u[(j - 1) * nx + i] + u[(j + 1) * nx + i]));
      res[j * nx + i] = temp;
      p[j * nx + i] = temp;
    }
}

template <typename tpe>
inline size_t conjugateGradient(const tpe *const __restrict__ rhs,
                                tpe *__restrict__ u, tpe *__restrict__ res,
                                tpe *__restrict__ p, tpe *__restrict__ ap,
                                const size_t nx, const size_t ny,
                                const size_t maxIt) {

  constexpr auto blockSize_x = 32, blockSize_y = 16;
  dim3 blockSize(blockSize_x, blockSize_y);
  int smcount = 0;
  hipDeviceGetAttribute(&smcount, hipDeviceAttributeMultiprocessorCount, 0);
  dim3 numBlocks(smcount, 10);

  // initialization
  tpe initResSq = (tpe)0;

  checkCudaError(hipMemPrefetchAsync(res, sizeof(tpe) * nx * ny, 0));
  checkCudaError(hipMemPrefetchAsync(p, sizeof(tpe) * nx * ny, 0));
  checkCudaError(hipMemPrefetchAsync(rhs, sizeof(tpe) * nx * ny, 0));
  checkCudaError(hipMemPrefetchAsync(u, sizeof(tpe) * nx * ny, 0));
  residual_initp<tpe><<<numBlocks, blockSize>>>(res, p, rhs, u, nx, ny);

  // compute residual norm
  checkCudaError(
    hipMemPrefetchAsync(res, sizeof(tpe) * nx * ny, hipCpuDeviceId));
  for (size_t j = 1; j < ny - 1; ++j) {
    for (size_t i = 1; i < nx - 1; ++i) {
      initResSq += res[j * nx + i] * res[j * nx + i];
    }
  }

  tpe curResSq = initResSq;

  // main loop
  for (size_t it = 0; it < maxIt; ++it) {
    nvtx3::scoped_range loop{"main loop"};

    nvtxRangePushA("Ap");
    // compute A * p
    checkCudaError(hipMemPrefetchAsync(p, sizeof(tpe) * nx * ny, 0));
    checkCudaError(hipMemPrefetchAsync(ap, sizeof(tpe) * nx * ny, 0));
    cgAp<tpe><<<numBlocks, blockSize>>>(p, ap, nx, ny);
    checkCudaError(hipDeviceSynchronize());
    nvtxRangePop();

    nvtxRangePushA("alpha");
    tpe alphaNominator = curResSq;
    checkCudaError(
      hipMemPrefetchAsync(p, sizeof(tpe) * nx * ny, hipCpuDeviceId));
    checkCudaError(
      hipMemPrefetchAsync(ap, sizeof(tpe) * nx * ny, hipCpuDeviceId));
    tpe alphaDenominator = (tpe)0;
    for (size_t j = 1; j < ny - 1; ++j) {
      for (size_t i = 1; i < nx - 1; ++i) {
        alphaDenominator += p[j * nx + i] * ap[j * nx + i];
      }
    }
    tpe alpha = alphaNominator / alphaDenominator;
    nvtxRangePop();

    // update solution
    nvtxRangePushA("solution");
    checkCudaError(hipMemPrefetchAsync(p, sizeof(tpe) * nx * ny, 0));
    checkCudaError(hipMemPrefetchAsync(u, sizeof(tpe) * nx * ny, 0));
    cgUpdateSol<tpe><<<numBlocks, blockSize>>>(p, u, alpha, nx, ny);
    checkCudaError(hipDeviceSynchronize());
    nvtxRangePop();

    // update residual
    nvtxRangePushA("residual");
    checkCudaError(hipMemPrefetchAsync(ap, sizeof(tpe) * nx * ny, 0));
    checkCudaError(hipMemPrefetchAsync(res, sizeof(tpe) * nx * ny, 0));
    cgUpdateRes<tpe><<<numBlocks, blockSize>>>(ap, res, alpha, nx, ny);
    checkCudaError(hipDeviceSynchronize());
    nvtxRangePop();

    // compute residual norm
    nvtxRangePushA("resNorm");
    checkCudaError(
      hipMemPrefetchAsync(res, sizeof(tpe) * nx * ny, hipCpuDeviceId));
    tpe nextResSq = (tpe)0;
    for (size_t j = 1; j < ny - 1; ++j) {
      for (size_t i = 1; i < nx - 1; ++i) {
        nextResSq += res[j * nx + i] * res[j * nx + i];
      }
    }
    nvtxRangePop();

    // check exit criterion
    if (sqrt(nextResSq) <= 1e-12)
      return it;

    // if (0 == it % 100)
    //     std::cout << "    " << it << " : " << sqrt(nextResSq) << std::endl;

    // compute beta
    nvtxRangePushA("beta");
    tpe beta = nextResSq / curResSq;
    curResSq = nextResSq;
    nvtxRangePop();

    // update p
    nvtxRangePushA("p");
    checkCudaError(hipMemPrefetchAsync(res, sizeof(tpe) * nx * ny, 0));
    checkCudaError(hipMemPrefetchAsync(p, sizeof(tpe) * nx * ny, 0));
    cgUpdateP<<<numBlocks, blockSize>>>(beta, res, p, nx, ny);
    checkCudaError(hipDeviceSynchronize());
    nvtxRangePop();
  }

  return maxIt;
}

template <typename tpe>
inline int realMain(int argc, char *argv[]) {
  char *tpeName;
  size_t nx, ny, nItWarmUp, nIt;
  parseCLA_2d(argc, argv, tpeName, nx, ny, nItWarmUp, nIt);

  tpe *u;
  checkCudaError(hipMallocManaged(&u, sizeof(tpe) * nx * ny));
  tpe *rhs;
  checkCudaError(hipMallocManaged(&rhs, sizeof(tpe) * nx * ny));

  // init
  initConjugateGradient(u, rhs, nx, ny);

  checkCudaError(hipMemPrefetchAsync(u, sizeof(tpe) * nx * ny, 0));
  checkCudaError(hipMemPrefetchAsync(rhs, sizeof(tpe) * nx * ny, 0));

  tpe *res;
  checkCudaError(hipMallocManaged(&res, sizeof(tpe) * nx * ny));
  tpe *p;
  checkCudaError(hipMallocManaged(&p, sizeof(tpe) * nx * ny));
  tpe *ap;
  checkCudaError(hipMallocManaged(&ap, sizeof(tpe) * nx * ny));

  checkCudaError(hipMemset(res, 0, sizeof(tpe) * nx * ny));
  checkCudaError(hipMemset(p, 0, sizeof(tpe) * nx * ny));
  checkCudaError(hipMemset(ap, 0, sizeof(tpe) * nx * ny));

  // warm-up
  nItWarmUp = conjugateGradient(rhs, u, res, p, ap, nx, ny, nItWarmUp);

  // measurement
  auto start = std::chrono::steady_clock::now();

  nIt = conjugateGradient(rhs, u, res, p, ap, nx, ny, nIt);
  std::cout << "  CG steps:      " << nIt << std::endl;

  auto end = std::chrono::steady_clock::now();

  printStats<tpe>(end - start, nIt, nx * ny, tpeName, 8 * sizeof(tpe), 15);

  checkCudaError(
    hipMemPrefetchAsync(u, sizeof(tpe) * nx * ny, hipCpuDeviceId));
  checkCudaError(
    hipMemPrefetchAsync(rhs, sizeof(tpe) * nx * ny, hipCpuDeviceId));

  // check solution
  checkSolutionConjugateGradient(u, rhs, nx, ny);

  checkCudaError(hipFree(res));
  checkCudaError(hipFree(p));
  checkCudaError(hipFree(ap));

  checkCudaError(hipFree(u));
  checkCudaError(hipFree(rhs));

  return 0;
}

int main(int argc, char *argv[]) {
  if (argc < 2) {
    std::cout << "Missing type specification " << std::endl;
    return 1;
  }

  std::string tpeName(argv[1]);

  if ("float" == tpeName)
    return realMain<float>(argc, argv);
  if ("double" == tpeName)
    return realMain<double>(argc, argv);

  std::cout << "Invalid type specification (" << argv[1]
            << "); supported types are" << std::endl;
  std::cout << "  float, double" << std::endl;
  return -1;
}
